#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008-2015, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */


#include "include/common.h"
#include "common.cuh"
#include "blocksync.cuh"

#include "include/histogram.h"


const unsigned int TAG_SHIFT = (32 - LOG2_WARP_SIZE);
const unsigned int TAG_MASK = (1U << TAG_SHIFT) - 1;

typedef volatile unsigned int histogram_t;

inline __device__ void addToBin(histogram_t *s_WarpHist, unsigned int data, unsigned int threadTag)
{
	unsigned int count;
	do {
		count = s_WarpHist[data] & TAG_MASK;
		count = threadTag | (count + 1);
		s_WarpHist[data] = count;
	} while (s_WarpHist[data] != count);
}


template <int BinCount>
inline __device__ void histogram1(unsigned int count,
	const float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms,
	histogram_t* s_Hist
)
{
	const unsigned int BlockSize = blockDim.x;
	const unsigned int WarpsPerBlock = (BlockSize >> LOG2_WARP_SIZE);

	const unsigned int idx = threadIdx.x;

	const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
	histogram_t* s_WarpHist = s_Hist + warpIdx * BinCount;

	//Clear shared memory storage for current threadblock before processing
	#pragma unroll
	for(unsigned int i = 0; i < (BinCount >> LOG2_WARP_SIZE); i++) {
	   s_Hist[idx + i * BlockSize] = 0;
	}

	__syncthreads();

	const unsigned int tag = (idx & (WARP_SIZE-1)) << TAG_SHIFT;

	for(unsigned int pos = (BlockSize*blockIdx.x + idx); pos < count; pos += BlockSize*gridDim.x)
	{
		float data = g_data[pos];
		if (data >= dataMin && data < dataMax)
		{
			unsigned int bin = (data - dataMin)*BinCount/(dataMax - dataMin);
			addToBin(s_WarpHist, bin, tag);
		}
	}

	//Merge per-warp histograms into per-block and write to global memory
	__syncthreads();
	if (idx < BinCount)
	{
		unsigned int sum = 0;

		for(unsigned int i = 0; i < WarpsPerBlock; i++)
			sum += s_Hist[idx + i * BinCount] & TAG_MASK;

		g_tmpHistograms[blockIdx.x * BinCount + idx] = sum;
	}
}

template <int BinCount>
inline __device__ void histogram2(
	const float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms,
	histogram_t* s_Hist, unsigned int gridSize
)
{
	const unsigned int idx = threadIdx.x;

	if (idx < BinCount)
	{
		s_Hist[idx] = 0;
		for (unsigned int i = 0; i < gridSize; ++i)
		{
			s_Hist[idx] += g_tmpHistograms[i*BinCount + idx];
		}
	}
	__syncthreads();

	//build CDF using prefix sum
	int pout = 0;
	int pin = 1;

	#pragma unroll
	for (int offset = 1; offset < BinCount; offset *= 2)
	{
		pout = 1 - pout;
		pin  = 1 - pout;

		if (idx < BinCount)
		{
			s_Hist[pout*BinCount + idx] = s_Hist[pin*BinCount + idx];
			if (idx >= offset)
				s_Hist[pout*BinCount + idx] += s_Hist[pin*BinCount + idx - offset];
#ifdef APEX_TEST
			g_tmpHistograms[pout*BinCount + idx] = s_Hist[pout*BinCount + idx];
#endif
		}

		__syncthreads();
	}

	if (idx == 0)
	{
		//unsigned int bound = g_bound[0];
		histogram_t* arr = s_Hist + pout*BinCount;
		
		//do binary search in CDF
		unsigned int beg = 0;
		unsigned int end = BinCount;
		while (beg < end)
		{
			unsigned int mid = beg + ((end - beg) >> 1);
			if (bound > arr[mid]) beg = mid + 1; else end = mid;
		}
		
		//g_dataMin[0] = dataMin + float(beg) * (dataMax - dataMin) / BinCount;
		//g_dataMax[0] = dataMin + float(beg+1) * (dataMax - dataMin) / BinCount;

		//assert( arr[beg] >= bound );
		g_boundParams[0] = bound - ((beg > 0) ? arr[beg-1] : 0);
		g_boundParams[1] = beg;
	}
}

SYNC_KERNEL_BEG(histogramSyncKernel, unsigned int _threadCount,
	const float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms
)
	extern __shared__ histogram_t s_Hist[]; /* size = [BinCount * WarpsPerBlock] */

	histogram1<HISTOGRAM_BIN_COUNT>(_threadCount, g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist);
	if (threadIdx.x < HISTOGRAM_BIN_COUNT)
	{
		__threadfence();
	}

	BLOCK_SYNC_BEGIN()

	histogram2<HISTOGRAM_BIN_COUNT>(g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist, gridDim.x);

	BLOCK_SYNC_END()

SYNC_KERNEL_END()

BOUND_KERNEL_BEG(histogramKernel,
	float *g_data, unsigned int bound, float dataMin, float dataMax, unsigned int* g_boundParams, unsigned int* g_tmpHistograms,
	unsigned int phase, unsigned int gridSize
)
	extern __shared__ histogram_t s_Hist[]; /* size = [BinCount * WarpsPerBlock] */

	if (phase == 1)
	{
		histogram1<HISTOGRAM_BIN_COUNT>(_threadCount, g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist);
	}
	else
	{
		histogram2<HISTOGRAM_BIN_COUNT>(g_data, bound, dataMin, dataMax, g_boundParams, g_tmpHistograms, s_Hist, gridSize);
	}

BOUND_KERNEL_END()
