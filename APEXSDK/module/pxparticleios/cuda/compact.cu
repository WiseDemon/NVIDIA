#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008-2015, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */


#include "include/common.h"
#include "common.cuh"

#include "include/compact.h"


inline __device__ unsigned int fetchHoleScan(unsigned int index, unsigned int& flag)
{
	const unsigned int holeScan = tex1Dfetch(KERNEL_TEX_REF(CompactScanSum), index);
	flag = (holeScan >> HOLE_SCAN_FLAG_BIT);
	return (holeScan & HOLE_SCAN_MASK); //inclusive
}
inline __device__ unsigned int fetchOutputScan(unsigned int index, unsigned int start, unsigned int holesBeforeStart, unsigned int& flag)
{
	unsigned int holeScan = fetchHoleScan(index, flag);
	if (index < start)
	{
		return holeScan; //inclusive
	}
	else
	{
		flag ^= 1;
		const unsigned int nonHoleScan = (index + 1) - holeScan; //inclusive
		const unsigned int nonHolesBeforeStart = start - holesBeforeStart;
		return holesBeforeStart + (nonHoleScan - nonHolesBeforeStart); //inclusive
	}
}

BOUND_KERNEL_BEG(compactKernel,
	unsigned int targetCount, unsigned int totalCount, unsigned int injectorCount, unsigned int* g_outIndices, unsigned int* g_outCount, unsigned int* g_injCounters
)
	const unsigned int start = targetCount;

	__shared__ unsigned int holesBeforeStart;
	if (threadIdx.x == 0) {
		unsigned int flag;
		holesBeforeStart = (start > 0) ? fetchHoleScan(start-1, flag) : 0;
		if (blockIdx.x == 0) {
			g_outCount[0] = holesBeforeStart;
		}
	}
	__syncthreads();

	{
		const unsigned int idx = threadIdx.x;
		const unsigned int warpIdx = (idx >> LOG2_WARP_SIZE);
		const unsigned int idxInWarp = idx & (WARP_SIZE-1);

		const unsigned int CountPerBlock = (totalCount + gridDim.x-1) / gridDim.x;

		const unsigned int DataWarpsPerBlock = (CountPerBlock + WARP_SIZE-1) / WARP_SIZE;
		const unsigned int WarpBorder = DataWarpsPerBlock % WarpsPerBlock;
		const unsigned int WarpFactor = DataWarpsPerBlock / WarpsPerBlock;

		const unsigned int WarpSelect = (warpIdx < WarpBorder) ? 1 : 0;
		const unsigned int WarpCount = WarpFactor + WarpSelect;
		const unsigned int WarpOffset = warpIdx * WarpCount + WarpBorder * (1 - WarpSelect);

		const unsigned int blockBeg = blockIdx.x * CountPerBlock;
		const unsigned int blockEnd = min(blockBeg + CountPerBlock, totalCount);

		const unsigned int warpBeg = blockBeg + (WarpOffset << LOG2_WARP_SIZE);
		const unsigned int warpEnd = min(warpBeg + (WarpCount << LOG2_WARP_SIZE), blockEnd);

		const unsigned int Log2BufferSize = (LOG2_WARP_SIZE + 1);
		const unsigned int BufferSize = (1 << Log2BufferSize);

		extern __shared__ volatile unsigned int sdata[]; /* size = [WarpsPerBlock * WARP_SIZE] */
		volatile unsigned int* buffer = sdata + (WarpsPerBlock << LOG2_WARP_SIZE); /* size = [WarpsPerBlock * BufferSize] */

		__shared__ volatile unsigned int outputBeg[MAX_WARPS_PER_BLOCK];  /* size = [WarpsPerBlock] */

		if (warpBeg < warpEnd)
		{
			if (idxInWarp == 0) {
				unsigned int flag;
				outputBeg[warpIdx] = (warpBeg > 0) ? fetchOutputScan(warpBeg-1, start, holesBeforeStart, flag) : 0;
			}

			unsigned int bufferBeg = outputBeg[warpIdx] & (WARP_SIZE-1);
			unsigned int bufferEnd = bufferBeg;

			if (idxInWarp == 0) {
				outputBeg[warpIdx] &= ~(WARP_SIZE-1);
			}

			for (unsigned int i = warpBeg; i < warpEnd; i += WARP_SIZE)
			{
				unsigned int inputPos = i + idxInWarp;
				if (inputPos < warpEnd)
				{
					unsigned int flag;
					unsigned int outputPos = fetchOutputScan(inputPos, start, holesBeforeStart, flag);
					unsigned int bufferPos = outputPos - outputBeg[warpIdx];

					sdata[idx] = bufferPos;
					if (flag)
					{
						bufferPos -= 1; //inclusive -> exclusive
						bufferPos += (bufferBeg & WARP_SIZE);
						bufferPos &= (BufferSize-1);

						buffer[(warpIdx << Log2BufferSize) + bufferPos] = inputPos;
					}
				}

				unsigned int endOfWarp = (min(i + WARP_SIZE, warpEnd)-1 - warpBeg) & (WARP_SIZE-1);
				bufferEnd = (bufferBeg & WARP_SIZE) + sdata[(warpIdx << LOG2_WARP_SIZE) + endOfWarp];
				bufferEnd &= (BufferSize-1);

				if ((bufferBeg & WARP_SIZE) != (bufferEnd & WARP_SIZE))
				{
					if (idxInWarp >= (bufferBeg & (WARP_SIZE-1)) ) {
						g_outIndices[outputBeg[warpIdx] + idxInWarp] = buffer[(warpIdx << Log2BufferSize) + (bufferBeg & WARP_SIZE) + idxInWarp];
					}
					bufferBeg = (bufferEnd & WARP_SIZE);
					if (idxInWarp == 0) {
						outputBeg[warpIdx] += WARP_SIZE;
					}
				}
			}

			if ( idxInWarp >= (bufferBeg & (WARP_SIZE-1)) && idxInWarp < (bufferEnd & (WARP_SIZE-1)) ) {
				g_outIndices[outputBeg[warpIdx] + idxInWarp] = buffer[(warpIdx << Log2BufferSize) + (bufferBeg & WARP_SIZE) + idxInWarp];
			}
		}
	}

	if (injectorCount > HISTOGRAM_SIMULATE_BIN_COUNT)
	{
		for (physx::PxU32 pos = BlockSize*blockIdx.x + threadIdx.x; pos < injectorCount; pos += BlockSize*gridDim.x)
		{
			g_injCounters[ pos ] = 0;
		}
	}
BOUND_KERNEL_END()
